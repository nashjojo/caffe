#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void RmseLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  num_rating_ = bottom[2]->cpu_data()[0];
  int count = bottom[0]->count();
  CHECK_LE(num_rating_, count) << "assigned rating length exceed boundary.";
  caffe_gpu_sub(
      num_rating_,
      bottom[0]->gpu_data(),
      bottom[1]->gpu_data(),
      diff_.mutable_gpu_data());
  if (bias_!=0) {
    caffe_gpu_add_scalar(num_rating_, bias_, diff_.mutable_gpu_data());
  }
  Dtype dot;
  caffe_gpu_dot(num_rating_, diff_.gpu_data(), diff_.gpu_data(), &dot);
  // Dtype loss = dot / bottom[0]->num() / Dtype(2);
  Dtype loss = sqrt(dot / num_rating_); // rmse, temp for movielens.
  (*top)[0]->mutable_cpu_data()[0] = loss;
  // LOG(INFO) << "loss:" << loss << " num_rating_:" << num_rating_;
}

template <typename Dtype>
void RmseLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      // LOG(INFO) << "propagate_down[i]:" << i << " top[0]->cpu_diff()[0]:" << top[0]->cpu_diff()[0]; // top[0]->cpu_diff()[0] is set to 1 by net that's the total loss of all instance.
      const Dtype sign = (i == 0) ? 1 : -1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] / num_rating_;
      caffe_gpu_axpby(
          num_rating_,                        // actual number of rating
          alpha,                              // alpha
          diff_.gpu_data(),                   // a
          Dtype(0),                           // beta
          (*bottom)[i]->mutable_gpu_diff());  // b
    }
  }
}

INSTANTIATE_CLASS(RmseLossLayer);

}  // namespace caffe
