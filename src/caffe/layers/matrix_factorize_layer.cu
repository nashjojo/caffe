#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void MatrixFactorizeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  if (force_cpu_) {
    return Forward_cpu(bottom, top);
  }

  // LOG(INFO) << "Forward_gpu";
  const Dtype* user_feature = this->blobs_[0]->gpu_data();
  const Dtype* item_feature = this->blobs_[1]->gpu_data();
  const Dtype* item_feature_img = bottom[0]->gpu_data();
  const Dtype* itact_data_ = bottom[1]->cpu_data();				// must use cpu
  const Dtype* itact_count_ = bottom[2]->cpu_data();			// must use cpu
  const Dtype* global_bias = this->blobs_[2]->cpu_data(); // must use cpu
  // user feature buffer
  Dtype* user_feature_buf = user_feature_buffer_.mutable_gpu_data();
  Dtype* item_feature_buf = item_feature_mixed_.mutable_gpu_data();
  Dtype* itact_pred_ = (*top)[0]->mutable_gpu_data();

  int item_offset = 0, rating_size = 0;
  int item_real_id = 0, userid = 0, rating_idx = 0;
  // relative itemid_version
  for (int itemid = 0; itemid < itact_item_; ++itemid ) {
    item_offset = itact_count_[itemid*2];
    rating_size = itact_count_[itemid*2+1];
    item_real_id = itact_data_[item_offset*2];
    caffe_copy(num_latent_, item_feature + item_real_id*num_latent_, item_feature_buf + itemid*num_latent_);
    caffe_gpu_axpby(num_latent_, Dtype(1.0), item_feature_img + itemid*num_latent_, Dtype(1.0), item_feature_buf + itemid*num_latent_);

    // LOG(INFO) << "itemid:" << itemid << " offset:" << item_offset << " rating_size: " << rating_size;
    for (int rating_cnt = 0; rating_cnt < rating_size; ++rating_cnt) {
      rating_idx = item_offset + rating_cnt;
      userid = static_cast<int>(itact_data_[rating_idx*2+1]);
      caffe_copy(num_latent_, user_feature + userid*num_latent_, user_feature_buf + rating_cnt*num_latent_);
      // LOG(INFO) << "itemid:" << itemid << " userid:" << userid;
    }
    caffe_gpu_gemv(CblasNoTrans, rating_size, num_latent_,
      Dtype(1.0), user_feature_buf, item_feature_buf + itemid*num_latent_, Dtype(0.),
      itact_pred_ + item_offset);
  }
  if (bias_term_) {
    caffe_gpu_add_scalar(num_rating_, global_bias[0], itact_pred_);
  }
  // set the extra space in itact_pred_ to 0
  if (num_rating_ < max_rating_size_) {
    int extra_length = max_rating_size_ - num_rating_;
    caffe_gpu_set(extra_length, Dtype(0.), itact_pred_ + num_rating_);
  }
  // checking prediction
  // const Dtype* itact_pred_cpu_ = (*top)[0]->cpu_data();
  // LOG(INFO) << "Global bias " << global_bias[0];
  // for (int j = 0; j < num_rating_; j++) {
  //   if (itact_pred_cpu_[j] > -2) {
  //     std::cout << j << ":" << itact_pred_cpu_[j] << "\t";
  //   }
  // } 
  // std::cout << std::endl;
}

template <typename Dtype>
void MatrixFactorizeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  if (force_cpu_) {
    return Backward_cpu(top, propagate_down, bottom);
  }
  
  // LOG(INFO) << "Backward_gpu";

  // checking loss
  // const Dtype* rating_diff = top[0]->cpu_diff();
  // LOG(INFO) << "loss ";
  // for (int j = 0; j < num_rating_; j++) {
  //   if (abs(rating_diff[j]) > 5e-7) {
  //     std::cout << j << ":" << rating_diff[j] << "\t";
  //   }
  // } 
  // std::cout << std::endl;

  Backward_User_gpu(top, propagate_down, bottom);
  Backward_Item_gpu(top, propagate_down, bottom);
  Backward_Item_img_gpu(top, propagate_down, bottom);
  // update global bias. 
  if (bias_term_ && this->param_propagate_down_[2]) {
    const Dtype* rating_diff = top[0]->gpu_diff();
    Dtype* bias_diff = this->blobs_[2]->mutable_gpu_diff();
    // Gradient with respect to bias
    caffe_gpu_gemv<Dtype>(CblasNoTrans, 1, num_rating_, Dtype(1.), rating_diff,
        bias_multiplier_.gpu_data(), Dtype(0.),
        bias_diff);
  }
}

template <typename Dtype>
void MatrixFactorizeLayer<Dtype>::Backward_User_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
	// return Backward_User_cpu(top, propagate_down, bottom);
  // LOG(INFO) << "Backward_User_gpu";
  // bp diff to user feature in blob_[0]
  if (this->param_propagate_down_[0]) {
    const Dtype* rating_diff = top[0]->cpu_diff();  // must be cpu
    Dtype* user_feature_diff = this->blobs_[0]->mutable_gpu_diff(); // Target
    const Dtype* item_feature = item_feature_mixed_.gpu_data();
    const Dtype* itact_data_ = (*bottom)[1]->cpu_data();  // must be cpu
    Dtype* item_feature_buf = item_feature_buffer_.mutable_gpu_data();

    int userid = 0, itemid = 0, rating_idx = 0, rating_size = 0;
    Dtype loss = 0;
    // traverse each user, #user gemv
    map<int, vector<int> >::iterator iter;
    for (iter = user2itemid.begin(); iter!=user2itemid.end(); ++iter) {
      userid = iter->first;
      vector<int>& ratingset = iter->second;
      rating_size = ratingset.size();
      rating_buffer_.Reshape(1,1,1,rating_size);
      Dtype* rating_buf = rating_buffer_.mutable_cpu_data();

      // std::cout << "userid:" << userid << " rating_size:" << rating_size << std::endl;
      for (int i = 0; i < rating_size; ++i) {
        rating_idx = ratingset[i];
        itemid = itemid2relative_id[static_cast<int>(itact_data_[rating_idx*2])]; // relative itemid
        // fill buffer
        loss = rating_diff[rating_idx];
        rating_buf[i] = loss;
        caffe_copy(num_latent_, item_feature + itemid*num_latent_, item_feature_buf + i*num_latent_);
        // std::cout << "rating_idx:" << rating_idx << " itemid:" << itemid << " loss:" << loss << std::endl;
      }
      caffe_gpu_gemv(CblasTrans, rating_size, num_latent_,
        Dtype(1.0) / rating_size, item_feature_buf, rating_buffer_.gpu_data(), Dtype(0.),
        user_feature_diff + userid*num_latent_);

      // LOG(INFO) << "item_feature_buf";
      // for (int i = 0; i < rating_size; ++i) {
      //   for (int j = 0; j < num_latent_; ++j) {
      //     std::cout << item_feature_buf[i*num_latent_ + j] << "\t";
      //   }
      //   std::cout << std::endl;
      // }

      // LOG(INFO) << "loss";
      // for (int i = 0; i < rating_size; i++) {
      //   std::cout << rating_buf[i] << "\t";
      // }
      // std::cout << std::endl;

      // LOG(INFO) << "user_feature_diff";
      // for (int i = 0; i < num_latent_; i++) {
      //   std::cout << user_feature_diff[userid*num_latent_ + i] << "\t";
      // }
      // std::cout << std::endl;
    }
  }
}

template <typename Dtype>
void MatrixFactorizeLayer<Dtype>::Backward_Item_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
	// return Backward_Item_cpu(top, propagate_down, bottom);
  // LOG(INFO) << "Backward_Item_gpu";
  // bp diff to item feature in blobs_[1]
  if (this->param_propagate_down_[1]) {
    // LOG(INFO) << "Backward_Item_gpu begin";
    if (!gen_item_diff_) {
      // LOG(INFO) << "Calculating";
      const Dtype* rating_diff = top[0]->gpu_diff();
      Dtype* item_feature_diff = this->blobs_[1]->mutable_gpu_diff(); // Target
      const Dtype* user_feature = this->blobs_[0]->gpu_data(); 
      const Dtype* itact_data_ = (*bottom)[1]->cpu_data();  // must be cpu
      const Dtype* itact_count_ = (*bottom)[2]->cpu_data(); // must be cpu
      Dtype* user_feature_buf = user_feature_buffer_.mutable_gpu_data();

      int item_offset = 0, rating_size = 0;
      int item_real_id = 0, userid = 0, rating_idx = 0;
      // relative itemid_version
      for (int itemid = 0; itemid < itact_item_; ++itemid ) {
        item_offset = itact_count_[itemid*2];
        rating_size = itact_count_[itemid*2+1];
        item_real_id = itact_data_[item_offset*2];
        // loss is inherently continuous
        // int item_real_id = static_cast<int>(itact_data_[item_offset*2]);
        // std::cout << "itemid:" << itemid << " item_real_id:" << item_real_id << " offset:" << item_offset << " rating_size: " << rating_size << std::endl;;
        for (int rating_cnt = 0; rating_cnt < rating_size; ++rating_cnt) {
          rating_idx = item_offset + rating_cnt; // simple mapping
          userid = static_cast<int>(itact_data_[rating_idx*2+1]);
          caffe_copy(num_latent_, user_feature + userid*num_latent_, user_feature_buf + rating_cnt*num_latent_);
          // std::cout << "rating_idx:" << rating_idx  << " itemid:" << itemid << " userid:" << userid << std::endl;
        }
        caffe_gpu_gemv(CblasTrans, rating_size, num_latent_,
          (Dtype(1.0) / 2) / rating_size, user_feature_buf, rating_diff + item_offset, Dtype(0.),
          item_feature_diff + item_real_id*num_latent_);

        gen_item_diff_ = true; // indicate we have computed item diff

        // LOG(INFO) << "user_feature_buf";
        // for (int i = 0; i < rating_size; ++i) {
        //   for (int j = 0; j < num_latent_; ++j) {
        //     std::cout << user_feature_buf[i*num_latent_ + j] << "\t";
        //   }
        //   std::cout << std::endl;
        // }

        // LOG(INFO) << "loss";
        // for (int i = 0; i < rating_size; i++) {
        //   std::cout << rating_diff[item_offset+i] << "\t";
        // }
        // std::cout << std::endl;

        // LOG(INFO) << "item_feature_diff";
        // for (int i = 0; i < num_latent_; i++) {
        //   std::cout << item_feature_diff[item_real_id*num_latent_ + i] << "\t";
        // }
        // std::cout << std::endl;
      }
    } else {
      // LOG(INFO) << "copying";
      // the diff is already calculated in bottom[0]->gpu_diff()
      const Dtype* item_feature_diff_source = (*bottom)[0]->gpu_diff();
      Dtype* item_feature_diff = this->blobs_[1]->mutable_gpu_diff(); // Target
      const Dtype* itact_data_ = (*bottom)[1]->cpu_data();
      const Dtype* itact_count_ = (*bottom)[2]->cpu_data();
      int item_offset = 0, item_real_id = 0;
      for (int itemid = 0; itemid < itact_item_; ++itemid ) {
        item_offset = itact_count_[itemid*2];
        item_real_id = itact_data_[item_offset*2];
        caffe_copy(num_latent_, item_feature_diff_source + itemid*num_latent_, item_feature_diff + item_real_id*num_latent_);
      } // ~ for
    } // ~ if (!gen_item_diff_)
  } // ~ if (this->param_propagate_down_[1])
  // LOG(INFO) << "done";
}

template <typename Dtype>
void MatrixFactorizeLayer<Dtype>::Backward_Item_img_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
  // return Backward_Item_cpu(top, propagate_down, bottom);
  // LOG(INFO) << "Backward_Item_img_gpu";
  // bp diff to item feature in bottom[0]
  if (propagate_down[0]) {
    // LOG(INFO) << "Backward_Item_img_gpu begin";
    if (!gen_item_diff_) {
      // LOG(INFO) << "Calculating";
      const Dtype* rating_diff = top[0]->gpu_diff();
      Dtype* item_feature_diff = (*bottom)[0]->mutable_gpu_diff(); // Target
      const Dtype* user_feature = this->blobs_[0]->gpu_data(); 
      const Dtype* itact_data_ = (*bottom)[1]->cpu_data();  // must be cpu
      const Dtype* itact_count_ = (*bottom)[2]->cpu_data(); // must be cpu
      Dtype* user_feature_buf = user_feature_buffer_.mutable_gpu_data();

      int item_offset = 0, rating_size = 0;
      int item_real_id = 0, userid = 0, rating_idx = 0;
      // relative itemid_version
      for (int itemid = 0; itemid < itact_item_; ++itemid ) {
        item_offset = itact_count_[itemid*2];
        rating_size = itact_count_[itemid*2+1];
        item_real_id = itact_data_[item_offset*2];
        // loss is inherently continuous
        // int item_real_id = static_cast<int>(itact_data_[item_offset*2]);
        // std::cout << "itemid:" << itemid << " item_real_id:" << item_real_id << " offset:" << item_offset << " rating_size: " << rating_size << std::endl;;
        for (int rating_cnt = 0; rating_cnt < rating_size; ++rating_cnt) {
          rating_idx = item_offset + rating_cnt; // simple mapping
          userid = static_cast<int>(itact_data_[rating_idx*2+1]);
          caffe_copy(num_latent_, user_feature + userid*num_latent_, user_feature_buf + rating_cnt*num_latent_);
          // std::cout << "rating_idx:" << rating_idx  << " itemid:" << itemid << " userid:" << userid << std::endl;
        }
        caffe_gpu_gemv(CblasTrans, rating_size, num_latent_,
          (Dtype(1.0) / 2) / rating_size, user_feature_buf, rating_diff + item_offset, Dtype(0.),
          item_feature_diff + itemid*num_latent_);

        gen_item_diff_ = true; // indicate we have computed item diff

        // LOG(INFO) << "user_feature_buf";
        // for (int i = 0; i < rating_size; ++i) {
        //   for (int j = 0; j < num_latent_; ++j) {
        //     std::cout << user_feature_buf[i*num_latent_ + j] << "\t";
        //   }
        //   std::cout << std::endl;
        // }

        // LOG(INFO) << "loss";
        // for (int i = 0; i < rating_size; i++) {
        //   std::cout << rating_diff[item_offset+i] << "\t";
        // }
        // std::cout << std::endl;

        // LOG(INFO) << "item_feature_diff";
        // for (int i = 0; i < num_latent_; i++) {
        //   std::cout << item_feature_diff[itemid*num_latent_ + i] << "\t";
        // }
        // std::cout << std::endl;
      }
    } else {
      // LOG(INFO) << "copying";
      // the diff is already calculated in blobs_[1]->gpu_diff()
      const Dtype* item_feature_diff_source = this->blobs_[1]->gpu_diff(); 
      Dtype* item_feature_diff = (*bottom)[0]->mutable_gpu_diff(); // Target
      const Dtype* itact_data_ = (*bottom)[1]->cpu_data();
      const Dtype* itact_count_ = (*bottom)[2]->cpu_data();
      int item_offset = 0, item_real_id = 0;
      for (int itemid = 0; itemid < itact_item_; ++itemid ) {
        item_offset = itact_count_[itemid*2];
        item_real_id = itact_data_[item_offset*2];
        caffe_copy(num_latent_, item_feature_diff_source + item_real_id*num_latent_, item_feature_diff + itemid*num_latent_);
      } // ~ for
    } // ~ if (!gen_item_diff_)
  } // ~ if (propagate_down[0])
  // LOG(INFO) << "done";
}

INSTANTIATE_CLASS(MatrixFactorizeLayer);

}  // namespace caffe
